#include "kernel_calls.cuh"

namespace kernel_calls
{
    // Helper function for using CUDA to add vectors in parallel.
    void processDataWithCuda(
        const char* data
        , const size_t data_length
        , const char* keywords
        , const size_t keywords_length
        , const size_t word_size
        , int* histogram)
    {
        int* dev_data{};
        int* dev_keywords{};
        int* dev_histogram{};
        hipError_t cudaStatus;

        try {
            // Choose which GPU to run on, change this on a multi-GPU system.
            cudaStatus = hipSetDevice(0);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

            // Allocate GPU buffers for three vectors (two input, one output)    .
            cudaStatus = hipMalloc((void**)&dev_data, data_length * word_size * sizeof(char));
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMalloc failed!");

            cudaStatus = hipMalloc((void**)&dev_keywords, keywords_length * word_size * sizeof(char));
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMalloc failed!");

            cudaStatus = hipMalloc((void**)&dev_histogram, keywords_length * sizeof(int));
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMalloc failed!");

            // Copy input vectors from host memory to GPU buffers.
            cudaStatus = hipMemcpy(dev_data, data, data_length * word_size * sizeof(char), hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMemcpy failed!");

            cudaStatus = hipMemcpy(dev_keywords, keywords, keywords_length * word_size * sizeof(char), hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMemcpy failed!");

            cudaStatus = hipMemcpy(dev_histogram, histogram, keywords_length * sizeof(int), hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMemcpy failed!");

            // Launch a kernel on the GPU with one thread for each element.
            //kernels::countWords << <1, size >> > (dev_data, data_length, dev_keywords, keywords_length, )

            // Check for any errors launching the kernel
            cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("addKernel launch failed: " + std::string(hipGetErrorString(cudaStatus)));

            // hipDeviceSynchronize waits for the kernel to finish, and returns
            // any errors encountered during the launch.
            cudaStatus = hipDeviceSynchronize();
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipDeviceSynchronize returned error code " + std::to_string(cudaStatus) + "after launching addKernel!");

            // Copy output vector from GPU buffer to host memory.
            cudaStatus = hipMemcpy(histogram, dev_histogram, keywords_length * sizeof(int), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMemcpy failed!");
        }
        catch (std::runtime_error& e) {
            hipFree(dev_data);
            hipFree(dev_keywords);
            hipFree(dev_histogram);
            throw std::runtime_error(e);
        }
    }

}