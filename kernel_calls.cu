#include "kernel_calls.cuh"

namespace kernel_calls
{
    constexpr size_t block_size{ 1024 };
    constexpr size_t grid_size{ 1024 };

    // Helper function for using CUDA to add vectors in parallel.
    void processDataWithCuda(
        const char* data
        , const size_t data_length
        , const char* keywords
        , int* histogram
        , float* compute_time
        , float* total_time)
    {
        char* dev_data{};
        char* dev_keywords{};
        int* dev_histogram{};
        hipError_t cudaStatus;
        hipEvent_t start;
        hipEvent_t start_c;
        hipEvent_t end;
        hipEvent_t end_c;

        try {
            // Choose which GPU to run on, change this on a multi-GPU system.
            cudaStatus = hipSetDevice(0);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

            cudaStatus = hipEventCreate(&start);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to create start event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventCreate(&start_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to create start_c event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventCreate(&end);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to create end event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventCreate(&end_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to create end_c event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventRecord(start);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to record start event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            // Allocate GPU buffers for three vectors (two input, one output)
            cudaStatus = hipMalloc((void**)&dev_data, data_length * general::word_size * sizeof(char));
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMalloc failed!");

            cudaStatus = hipMalloc((void**)&dev_keywords, general::keywords_length * general::word_size * sizeof(char));
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMalloc failed!");

            cudaStatus = hipMalloc((void**)&dev_histogram, general::keywords_length * sizeof(int));
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMalloc failed!");

            // Copy input vectors from host memory to GPU buffers.
            cudaStatus = hipMemcpy(dev_data, data, data_length * general::word_size * sizeof(char), hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMemcpy failed!");

            cudaStatus = hipMemcpy(dev_keywords, keywords, general::keywords_length * general::word_size * sizeof(char), hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMemcpy failed!");

            cudaStatus = hipMemcpy(dev_histogram, histogram, general::keywords_length * sizeof(int), hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMemcpy failed!");

            cudaStatus = hipEventRecord(start_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to record start_c event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            // Launch a kernel on the GPU with one thread for each element.
            kernels::countWords << <grid_size, block_size >> > (dev_data, data_length, dev_keywords, dev_histogram);

            // Check for any errors launching the kernel
            cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Kernel launch failed: " + std::string(hipGetErrorString(cudaStatus)));

            // hipDeviceSynchronize waits for the kernel to finish, and returns
            // any errors encountered during the launch.
            cudaStatus = hipDeviceSynchronize();
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipDeviceSynchronize returned error code " + std::to_string(cudaStatus) + "after launching addKernel!" + hipGetErrorString(cudaStatus));

            cudaStatus = hipEventRecord(end_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to record end_c event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            // Copy output vector from GPU buffer to host memory.
            cudaStatus = hipMemcpy(histogram, dev_histogram, general::keywords_length * sizeof(int), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMemcpy failed!");
            
            cudaStatus = hipEventRecord(end);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to record end event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventSynchronize(end_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to sync end_c event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventSynchronize(end);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to sync end event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventElapsedTime(total_time, start, end);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to take elapsed end event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventElapsedTime(compute_time, start_c, end_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to take elapsed end event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            hipFree(dev_data);
            hipFree(dev_keywords);
            hipFree(dev_histogram);
        }
        catch (std::runtime_error& e) {
            hipFree(dev_data);
            hipFree(dev_keywords);
            hipFree(dev_histogram);
            throw std::runtime_error(e);
        }
    }


    void processDataWithCudaPreprocess(
        const char* data
        , const size_t data_length
        , const char* keywords
        , int* histogram
        , float* compute_time
        , float* total_time)
    {
        char* dev_data{};
        char* dev_keywords{};
        int* dev_histogram{};
        hipError_t cudaStatus;
        hipEvent_t start;
        hipEvent_t start_c;
        hipEvent_t end;
        hipEvent_t end_c;

        try {
            // Choose which GPU to run on, change this on a multi-GPU system.
            cudaStatus = hipSetDevice(0);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

            cudaStatus = hipEventCreate(&start);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to create start event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventCreate(&start_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to create start_c event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventCreate(&end);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to create end event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventCreate(&end_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to create end_c event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventRecord(start);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to record start event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            // Allocate GPU buffers for three vectors (two input, one output)
            cudaStatus = hipMalloc((void**)&dev_data, data_length * general::word_size * sizeof(char));
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMalloc failed!");

            cudaStatus = hipMalloc((void**)&dev_keywords, general::keywords_length * general::word_size * sizeof(char));
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMalloc failed!");

            cudaStatus = hipMalloc((void**)&dev_histogram, general::keywords_length * sizeof(int));
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMalloc failed!");

            // Copy input vectors from host memory to GPU buffers.
            cudaStatus = hipMemcpy(dev_data, data, data_length * general::word_size * sizeof(char), hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMemcpy failed!");

            cudaStatus = hipMemcpy(dev_keywords, keywords, general::keywords_length * general::word_size * sizeof(char), hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMemcpy failed!");

            cudaStatus = hipMemcpy(dev_histogram, histogram, general::keywords_length * sizeof(int), hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMemcpy failed!");

            cudaStatus = hipEventRecord(start_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to record start_c event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            // Preprocess data
            // Launch lowercasing kernel on data
            kernels::lowerData << <grid_size, block_size >> > (dev_data, data_length);

            // Check for any errors launching the kernel
            cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Kernel launch failed: " + std::string(hipGetErrorString(cudaStatus)));

            // hipDeviceSynchronize waits for the kernel to finish, and returns
            // any errors encountered during the launch.
            cudaStatus = hipDeviceSynchronize();
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipDeviceSynchronize returned error code " + std::to_string(cudaStatus) + "after launching addKernel!" + hipGetErrorString(cudaStatus));

            // Remove exessive punctuation marks and clear data
            kernels::removeExcessives << <grid_size, block_size >> > (dev_data, data_length);

            // Check for any errors launching the kernel
            cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Kernel launch failed: " + std::string(hipGetErrorString(cudaStatus)));

            // hipDeviceSynchronize waits for the kernel to finish, and returns
            // any errors encountered during the launch.
            cudaStatus = hipDeviceSynchronize();
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipDeviceSynchronize returned error code " + std::to_string(cudaStatus) + "after launching addKernel!" + hipGetErrorString(cudaStatus));

            // Process data
            kernels::countWords << <grid_size, block_size >> > (dev_data, data_length, dev_keywords, dev_histogram);

            // Check for any errors launching the kernel
            cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Kernel launch failed: " + std::string(hipGetErrorString(cudaStatus)));

            // hipDeviceSynchronize waits for the kernel to finish, and returns
            // any errors encountered during the launch.
            cudaStatus = hipDeviceSynchronize();
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipDeviceSynchronize returned error code " + std::to_string(cudaStatus) + "after launching addKernel!" + hipGetErrorString(cudaStatus));

            cudaStatus = hipEventRecord(end_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to record end_c event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            // Copy output vector from GPU buffer to host memory.
            cudaStatus = hipMemcpy(histogram, dev_histogram, general::keywords_length * sizeof(int), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMemcpy failed!");

            cudaStatus = hipEventRecord(end);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to record end event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventSynchronize(end_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to sync end_c event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventSynchronize(end);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to sync end event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventElapsedTime(total_time, start, end);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to take elapsed end event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventElapsedTime(compute_time, start_c, end_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to take elapsed end event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            hipFree(dev_data);
            hipFree(dev_keywords);
            hipFree(dev_histogram);
        }
        catch (std::runtime_error& e) {
            hipFree(dev_data);
            hipFree(dev_keywords);
            hipFree(dev_histogram);
            throw std::runtime_error(e);
        }
    }

    
    void processDataWithCudaStreamsPreprocess(
        const char* data
        , const size_t data_length
        , const char* keywords
        , int* histogram
        , float* compute_time
        , float* total_time)
    {
        char* host_data[n_streams]{};
        char* dev_data[n_streams]{};
        char* dev_keywords{};
        int* dev_histogram{};
        hipError_t cudaStatus;
        hipEvent_t start;
        hipEvent_t start_c;
        hipEvent_t end;
        hipEvent_t end_c;

        hipStream_t streams[n_streams];
        size_t stream_size{ data_length * general::word_size / n_streams };

        try {
            // Choose which GPU to run on, change this on a multi-GPU system.
            cudaStatus = hipSetDevice(0);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            
            cudaStatus = hipEventCreate(&start);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to create start event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventCreate(&start_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to create start_c event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventCreate(&end);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to create end event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventCreate(&end_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to create end_c event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventRecord(start);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to record start event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipMalloc((void**)&dev_keywords, general::keywords_length * general::word_size * sizeof(char));
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMalloc failed!");

            cudaStatus = hipMalloc((void**)&dev_histogram, general::keywords_length * sizeof(int));
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMalloc failed!");

            cudaStatus = hipMemcpy(dev_keywords, keywords, general::keywords_length * general::word_size * sizeof(char), hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMemcpy failed!");

            cudaStatus = hipMemcpy(dev_histogram, histogram, general::keywords_length * sizeof(int), hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMemcpy failed!");
            
            cudaStatus = hipEventRecord(start_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to record start_c event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            // Create and setup streams
            for (int i{}; i < n_streams; ++i) {
                cudaStatus = hipStreamCreate(&streams[i]);
                if (cudaStatus != hipSuccess)
                    throw std::runtime_error("Failed to create stream (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");
                
                cudaStatus = hipHostMalloc((void**)&host_data[i], stream_size * sizeof(char));
                if (cudaStatus != hipSuccess)
                    throw std::runtime_error("Failed to host-allocate data (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

                cudaStatus = hipMemcpyAsync(host_data[i], &data[i * stream_size], stream_size * sizeof(char), hipMemcpyHostToHost, streams[i]);
                if (cudaStatus != hipSuccess)
                    throw std::runtime_error("Failed to memcpy-async data (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

                cudaStatus = hipMallocAsync((void**)&dev_data[i], stream_size * sizeof(char), streams[i]);
                if (cudaStatus != hipSuccess)
                    throw std::runtime_error("Failed to device-allocate data (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

                cudaStatus = hipMemcpyAsync(dev_data[i], host_data[i], stream_size * sizeof(char), hipMemcpyHostToDevice, streams[i]);
                if (cudaStatus != hipSuccess)
                    throw std::runtime_error("Failed to memcpy-async data (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

                // Preprocess data
                // Launch lowercasing kernel on data
                kernels::lowerData << <grid_size / n_streams, block_size, 0, streams[i] >> > (dev_data[i], data_length / n_streams);

                // Check for any errors launching the kernel
                cudaStatus = hipGetLastError();
                if (cudaStatus != hipSuccess)
                    throw std::runtime_error("Kernel launch failed: " + std::string(hipGetErrorString(cudaStatus)));

                // hipDeviceSynchronize waits for the kernel to finish, and returns
                // any errors encountered during the launch.
                cudaStatus = hipStreamSynchronize(streams[i]);
                if (cudaStatus != hipSuccess)
                    throw std::runtime_error("hipDeviceSynchronize returned error code " + std::to_string(cudaStatus) + "after launching addKernel!" + hipGetErrorString(cudaStatus));

                // Remove exessive punctuation marks and clear data
                kernels::removeExcessives << <grid_size / n_streams, block_size, 0, streams[i] >> > (dev_data[i], data_length / n_streams);

                // Check for any errors launching the kernel
                cudaStatus = hipGetLastError();
                if (cudaStatus != hipSuccess)
                    throw std::runtime_error("Kernel launch failed: " + std::string(hipGetErrorString(cudaStatus)));

                // hipDeviceSynchronize waits for the kernel to finish, and returns
                // any errors encountered during the launch.
                cudaStatus = hipStreamSynchronize(streams[i]);
                if (cudaStatus != hipSuccess)
                    throw std::runtime_error("hipDeviceSynchronize returned error code " + std::to_string(cudaStatus) + "after launching addKernel!" + hipGetErrorString(cudaStatus));

                // Process data
                kernels::countWords << <grid_size / n_streams, block_size, general::keywords_length * (general::word_size + 1), streams[i] >> > (dev_data[i], data_length / n_streams, dev_keywords, dev_histogram);

                // Check for any errors launching the kernel
                cudaStatus = hipGetLastError();
                if (cudaStatus != hipSuccess)
                    throw std::runtime_error("Kernel launch failed: " + std::string(hipGetErrorString(cudaStatus)));

            }

            // hipDeviceSynchronize waits for the kernel to finish, and returns
            // any errors encountered during the launch.
            cudaStatus = hipDeviceSynchronize();
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipDeviceSynchronize returned error code " + std::to_string(cudaStatus) + "after launching addKernel!" + hipGetErrorString(cudaStatus));

            cudaStatus = hipEventRecord(end_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to record end_c event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            // Copy output vector from GPU buffer to host memory.
            cudaStatus = hipMemcpy(histogram, dev_histogram, general::keywords_length * sizeof(int), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("hipMemcpy failed!");

            cudaStatus = hipEventRecord(end);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to record end event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventSynchronize(end_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to sync end_c event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventSynchronize(end);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to sync end event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventElapsedTime(total_time, start, end);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to take elapsed end event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            cudaStatus = hipEventElapsedTime(compute_time, start_c, end_c);
            if (cudaStatus != hipSuccess)
                throw std::runtime_error("Failed to take elapsed end event (error code: " + std::string(hipGetErrorString(cudaStatus)) + ")!");

            hipFree(dev_keywords);
            hipFree(dev_histogram);
            for (int i{}; i < n_streams; ++i) {
                hipFree(host_data[i]);
                hipFree(dev_data[i]);
            }
        }
        catch (std::runtime_error& e) {
            hipFree(dev_keywords);
            hipFree(dev_histogram);
            for (int i{}; i < n_streams; ++i) {
                hipFree(host_data[i]);
                hipFree(dev_data[i]);
            }
            throw std::runtime_error(e);
        }
    }
}