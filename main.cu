// Windows
#include <windows.h>

// C++
#include <iostream>

// CUDA
#include "hip/hip_runtime.h"
#include ""

// Package
#include "kernel_calls.cuh"
#include "general.hpp"


int main()
{
    constexpr size_t word_size{ 32 }; // bytes
    // Reading keywords
    constexpr size_t keywords_length{ 1'024 }; // words
    std::string keyword_file{ "./data/google-10000-english-no-swears.txt" };
    char* keywords{ new char[keywords_length * word_size] };

    general::readWordFile(keyword_file, keywords, keywords_length, word_size);

    // Reading data
    constexpr size_t small_data_length{ 131'072 }; // words
    std::string small_data_file{ "./data/small.txt" };
    
    constexpr size_t medium_data_length{ 393'216 }; // words
    std::string medium_data_file{ "./data/small.txt" };
    
    constexpr size_t large_data_length{ 786'432 }; // words
    std::string large_data_file{ "./data/small.txt" };
    
    char* data{ new char[small_data_length * word_size] };

    general::readWordFile(small_data_file, data, small_data_length, word_size);

    hipError_t cudaStatus;

    // Add vectors in parallel.
    //hipError_t cudaStatus = kernel_calls::addWithCuda(c, a, b, arraySize);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
