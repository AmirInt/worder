// Windows
#include <windows.h>

// C++
#include <iostream>

// CUDA
#include "hip/hip_runtime.h"
#include ""

// Package
#include "kernel_calls.cuh"
#include "general.hpp"


int main()
{
    constexpr size_t word_size{ 32 }; // bytes
    // Reading keywords
    constexpr size_t keywords_length{ 1'024 }; // words
    std::string keyword_file{ "./data/google-10000-english-no-swears.txt" };
    char* keywords{ new char[keywords_length * word_size] };

    general::readWordFile(keyword_file, keywords, keywords_length, word_size);

    // Reading data
    constexpr size_t small_data_length{ 131'072 }; // words
    std::string small_data_file{ "./data/small.txt" };
    
    constexpr size_t medium_data_length{ 393'216 }; // words
    std::string medium_data_file{ "./data/small.txt" };
    
    constexpr size_t large_data_length{ 786'432 }; // words
    std::string large_data_file{ "./data/small.txt" };
    
    char* data{ new char[small_data_length * word_size] };

    general::readWordFile(small_data_file, data, small_data_length, word_size);

    // Histogram
    int* histogram{ new int[keywords_length] };

    hipError_t cudaStatus;

    // Process data in parallel.
    kernel_calls::processDataWithCuda(
        data
        , small_data_length
        , keywords
        , keywords_length
        , word_size
        , histogram);
    
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
        throw std::runtime_error("hipDeviceReset failed!");

    return 0;
}
